#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>

#define BLOCK_SIZE 32

__global__ void matrix_transpose_naive(int *input, int *output, int n) {

	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	int index = indexY * n + indexX;
	int transposedIndex = indexX * n + indexY;

    // this has discoalesced global memory store  
	output[transposedIndex] = input[index];

	// this has discoalesced global memore load
	// output[index] = input[transposedIndex];
}

__global__ void matrix_transpose_shared(int *input, int *output, int n) {

	__shared__ int sharedMemory [BLOCK_SIZE] [BLOCK_SIZE];

	// global index	
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;

	// transposed global memory index
	int tindexX = threadIdx.x + blockIdx.y * blockDim.x;
	int tindexY = threadIdx.y + blockIdx.x * blockDim.y;

	// local index
	int localIndexX = threadIdx.x;
	int localIndexY = threadIdx.y;

	int index = indexY * n + indexX;
	int transposedIndex = tindexY * n + tindexX;

	// reading from global memory in coalesed manner and performing tanspose in shared memory
	sharedMemory[localIndexX][localIndexY] = input[index];

	__syncthreads();

	// writing into global memory in coalesed fashion via transposed data in shared memory
	output[transposedIndex] = sharedMemory[localIndexY][localIndexX];
}

//basically just fills the array with index.
void fill_array(int *data, int n) {
	for(int idx=0;idx<(n*n);idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int n) {
	printf("\n Original Matrix::\n");
	for(int idx=0;idx<(n*n);idx++) {
		if(idx%n == 0)
			printf("\n");
		printf(" %d ",  a[idx]);
	}
	printf("\n Transposed Matrix::\n");
	for(int idx=0;idx<(n*n);idx++) {
		if(idx%n == 0)
			printf("\n");
		printf(" %d ",  b[idx]);
	}
}

int main(void) {
	int *a, *b;
        int *d_a, *d_b; // device copies of a, b, c

	hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  for (int n = 1028; n <= 10280; n += 1028){  
    int size = n * n *sizeof(int);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); fill_array(a, n);
    b = (int *)malloc(size);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE,1);
    dim3 gridSize(n/BLOCK_SIZE,n/BLOCK_SIZE,1);

    for (int i = 0; i < 5; i++){
    hipEventRecord(start);
    // matrix_transpose_naive<<<gridSize,blockSize>>>(d_a,d_b,n);
    matrix_transpose_shared<<<gridSize,blockSize>>>(d_a,d_b,n);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float time = 0;
    hipEventElapsedTime(&time, start, end);
    printf("%d,%f,%d\n", n, time, i);
}
    free(a);
    free(b);
      hipFree(d_a);
    hipFree(d_b); 
  }

	return 0;
}